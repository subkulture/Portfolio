// C++ / CUDA Program to perform matrix maths using GPUs via CUDA


// Includes

#include <hip/hip_runtime.h>
#include <iostream>  // cout
#include <iomanip>  // setprecision
#include <stdlib.h>  // atoi
using namespace std;

//--------------------------------------------------------------------
// CUDA Kernel function to add the elements of two arrays on the GPU
//--------------------------------------------------------------------
__global__ // all kernels are preceded by __global__ keyword
void add(int n, // number of elements in an array
         float *A, // device pointer to array
         float *B, // device pointer to another array
         float *C) // device pointer to another array
{
    // determine thread ID within block
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // determine stride of loop (more elements in array than threads)
    int stride = blockDim.x * gridDim.x;

    // each thread loops over array elements in steps of grid size
    for (int i = index; i < n; i += stride){
        A[i] = B[i] + C[i];
    }
}

int main(int argc, char** argv)
{
    // Get N from command line argument
    int N = 1000000;
    if (argc > 1) N = atoi(argv[1]); // specify array size
    std::cout << "Running with N = " << N << std::endl;
    float *A, *B, *C; // pointers for arrays

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&A, N*sizeof(float));
    hipMallocManaged(&B, N*sizeof(float));
    hipMallocManaged(&C, N*sizeof(float));

    // initialize arrays on the host
    for(int i = 0; i < N; i++) A[i] = 0.0f;
    for(int i = 0; i < N; i++) B[i] = 0.1f;
    for(int i = 0; i < N; i++) C[i] = 0.2f;

    // Check the arithmetic on the CPU.
    double a;
    float b, c;
    b = 0.1f;
    c = 0.2f;
    a = b + c;
    std::cout << "a = " << std::setprecision(16) << a << std::endl;
    std::cout << "delta = " << std::setprecision(16) << 0.3 - a << std::endl;

    int num_threads_in_block = 256; // set number of threads (multiple of 32)
    int num_blocks = 32*2; // for 2 SMs, set a multiple blocks for each one

    // Run kernel on N elements on the GPU
    add<<<num_blocks, num_threads_in_block>>>(N, A, B, C);

    // cudaDeviceSynchronize call added here as the CPU continues through the rest of the
    // program so the printing to std::cout and cudaFree calls will operate on the arrays before the
    // GPU has finished doing its work.
    hipDeviceSynchronize();

    std::cout << "A[0] = " << std::setprecision(16) << double(A[0]) << std::endl;
    std::cout << "delta = " << std::setprecision(16) <<  0.3 - double(A[0]) << std::endl;

    hipFree(A);
    hipFree(B);
    hipFree(C);
}
